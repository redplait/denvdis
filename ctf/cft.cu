#include <hip/hip_runtime.h>
#include <stdio.h>

#define WARP_SIZE (1<<5)
// from https://github.com/abdimoallim/cuda-utils/blob/main/cutils.cuh
__device__ __forceinline__ int warp_reduce_sum(int val) {
  for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2) {
    val += __shfl_down_sync(0xFFFFFFFF, val, offset);
  }
  return val;
}

__device__ __forceinline__ int warp_reduce_min(int val) {
  for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2) {
    val = min(val, __shfl_down_sync(0xFFFFFFFF, val, offset));
  }
  return val;
}

__constant__ static const char seed[32] = {
 0x2f ^ 'a',
 0x2f ^ 'b',
 0x2f ^ 'c',
 0x2f ^ 'd',
 0x2f ^ 'e',
 0x2f ^ 'f',
 0x2f ^ 'g',
 0x2f ^ 'h',
 0x2f ^ 'i',
 0x2f ^ 'j',
 0x2f ^ 'k',
 0x2f ^ 'l',
 0x2f ^ 'm',
 0x2f ^ 'n',
 0x2f ^ 'o',
 0x2f ^ 'p',
 0x2f ^ 'r',
 0x2f ^ 's',
 0x2f ^ 't',
 0x2f ^ 'u',
 0x2f ^ 'v',
 0x2f ^ 'w',
 0x2f ^ 'x',
 0x2f ^ 'y',
 0x2f ^ 'z',
 0x2f ^ '0',
 0x2f ^ '1',
 0x2f ^ '2',
 0x2f ^ '3',
 0x2f ^ '4',
 0x2f ^ '5',
 0x2f ^ '6',
};

#include <stdint.h>

__global__ void machine_ids(uint32_t *out_buf)
{
  // all threadIdx will be replaced with ced to SR_MACHINE_ID_X
  uint32_t x = threadIdx.x;
  out_buf[0] = x;
  x = threadIdx.y;
  out_buf[1] = x;
  x = threadIdx.z;
  out_buf[2] = x;
  asm volatile("mov.u32 %0, %%smid;" : "=r"(x));
  out_buf[3] = x;
  out_buf[4] = 0x15;
}

__global__ void calc_hash(const char *s, int *res)
{
  int x = threadIdx.x;
  int v = (s[x] ^ seed[x] ^ 0x2f) ? 0 : 1;
  __syncthreads();
  int r = warp_reduce_min(v);
  if ( !x ) {
// printf("res %X\n", r);
    *res = r;
  }
}

#include <string>
#include <iostream>
// #include <cuda_helper.h>
#define checkCudaErrors(err) { \
if (err != hipSuccess) { \
 fprintf(stderr, "checkCudaErrors() API error = %04d \"%s\" from file <%s>, line %i.\n", \
 err, hipGetErrorString(err), __FILE__, __LINE__); \
 exit(-1); \
} \
}

// main
__host__ int main()
{
  std::string s; // = "abcdefghijklmnoprstuvwxyz0123456";
  std::cin >> s;
  if ( s.size() != 32 ) {
    printf("bad len of string\n");
    return 1;
  }
  uint32_t *card_id;
  // read card id - 4 * 4 = 16 bytes + 4 for test
  auto err = hipMalloc(&card_id, 20);
  checkCudaErrors(err);
  machine_ids<<<1,1>>>(card_id);
  err = hipDeviceSynchronize(); checkCudaErrors(err);
  uint32_t host_card_id[5];
  err = hipMemcpy(host_card_id, card_id, sizeof(host_card_id), hipMemcpyDeviceToHost); checkCudaErrors(err);
  // dump card id
  unsigned char *cid = (unsigned char *)host_card_id;
  for ( int i = 0; i < 20; i++ ) printf("%2.2X ", cid[i]);
  fputc('\n', stdout);
  hipFree(card_id);
  // rest
  char *d_c;
  int *d_i;
  err = hipMalloc(&d_c, 32); checkCudaErrors(err);
  err = hipMalloc(&d_i, sizeof(int)); checkCudaErrors(err);
  err = hipMemcpy(d_c, s.c_str(), 32, hipMemcpyHostToDevice); checkCudaErrors(err);
  calc_hash<<<1,32>>>(d_c, d_i);
  err = hipDeviceSynchronize(); checkCudaErrors(err);
  int res = 1;
  err = hipMemcpy(&res, d_i, sizeof(res), hipMemcpyDeviceToHost); checkCudaErrors(err);
  hipFree(d_c);
  hipFree(d_i);
  if ( res )
   printf("yes\n");
  else
   printf("no\n");
  return res;
}